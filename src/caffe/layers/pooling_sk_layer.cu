#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"


// (const int nthreads, const Dtype* bottom_data,
    // const int num, const int channels, const int height,
    // const int width, const int pooled_height, const int pooled_width,
    // const int kernel_h, const int kernel_w, const int ext_kernel_h, const int ext_kernel_w,
    // const int stride_h, const int stride_w, const int kstride_h, const int kstride_w, 
    // const int pad_h, const int pad_w, Dtype* top_data,
    // int* mask, Dtype* top_mask) 


namespace caffe {


// __global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    // const int num, const int channels, const int height, const int width, const int depth, 
	// const int pooled_height, const int pooled_width, const int pooled_depth, 
	// const int kernel_h, const int kernel_w, const int kernel_d,
    // const int ext_kernel_h, const int ext_kernel_w, const int ext_kernel_d,	
	// const int stride_h, const int stride_w, const int stride_d, 
	// const int kstride_h, const int kstride_w, const int kstride_d,
	// const int pad_h, const int pad_w, const int pad_d, Dtype* top_data,
    // int* mask, Dtype* top_mask)
template <typename Dtype>
__global__ void MaxPoolForward(const size_t nthreads, const Dtype* bottom_data,
    const size_t num, const size_t channels, const size_t height, const size_t width, const size_t depth, 
	const size_t pooled_height, const size_t pooled_width, const size_t pooled_depth, 
	const size_t kernel_h, const size_t kernel_w, const size_t kernel_d,
    const size_t ext_kernel_h, const size_t ext_kernel_w, const size_t ext_kernel_d,	
	const size_t stride_h, const size_t stride_w, const size_t stride_d, 
	const size_t kstride_h, const size_t kstride_w, const size_t kstride_d,
	const size_t pad_h, const size_t pad_w, const size_t pad_d, Dtype* top_data,
    int* mask, Dtype* top_mask)
	{
	typedef unsigned long long ulong;
  CUDA_KERNEL_LOOP(index, nthreads) {
	ulong pd = index % pooled_depth;
	ulong pw = (index / pooled_depth) % pooled_width;
    ulong ph = (index / pooled_depth / pooled_width ) % pooled_height;
    ulong c = (index / pooled_depth / pooled_width / pooled_height) % channels;
    ulong n = index / pooled_depth / pooled_width / pooled_height / channels;
    ulong hstart = ph * stride_h - pad_h;
    ulong wstart = pw * stride_w - pad_w;
	ulong dstart = pd * stride_d - pad_d;
    ulong hend = min(hstart + ext_kernel_h, (ulong)height);
    ulong wend = min(wstart + ext_kernel_w, (ulong)width);
	ulong dend = min(dstart + ext_kernel_d, (ulong)depth);
    hstart = max(hstart, (ulong)0);
    wstart = max(wstart, (ulong)0);
	dstart = max(dstart, (ulong)0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width * depth;
    for (size_t h = hstart; h < hend; h+=kstride_h ) {
      for (size_t w = wstart; w < wend; w+=kstride_w) {
	    for (size_t d = dstart; d < dend; d+=kstride_d){
		const size_t m_idx =  (h * width + w) * depth +d;
        if (bottom_data[m_idx] > maxval) {
          //maxidx = (h * width + w) * depth + d;
		  maxidx = static_cast<int>(m_idx);
          maxval = bottom_data[maxidx];
		  }
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}
  

template <typename Dtype>
__global__ void MaxPoolBackward_SK(const int nthreads, const Dtype* top_diff,
    const int* mask, const Dtype* top_mask, const int num, const int channels,
    const int height, const int width, const int depth, const int pooled_height,
    const int pooled_width, const int pooled_depth, 
	const int kernel_h, const int kernel_w, const int kernel_d, 
	const int ext_kernel_h, const int ext_kernel_w, const int ext_kernel_d, 
	const int stride_h, const int stride_w, const int stride_d,	
    const int pad_h, const int pad_w, const int pad_d, 
	const int k_stride_h, const int k_stride_w, const int k_stride_d,	
	Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
	int d = index % depth;
    int w = (index / depth) % width;
    int h = (index / depth / width) % height;
    int c = (index / depth / width / height) % channels;
    int n = index / depth / width / height / channels;
    // int phstart =
        // (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    // int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    // int pwstart =
        // (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    // int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
	
	// int pdstart =
        // (d + pad_d < kernel_d) ? 0 : (d + pad_d - kernel_d) / stride_d + 1;
    // int pdend = min((d + pad_d) / stride_d + 1, pooled_depth);
	
	
	int phstart =
        (h + pad_h < ext_kernel_h) ? 0 : (h + pad_h - ext_kernel_h) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart =
        (w + pad_w < ext_kernel_w) ? 0 : (w + pad_w - ext_kernel_w) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
	
	int pdstart =
        (d + pad_d < ext_kernel_d) ? 0 : (d + pad_d - ext_kernel_d) / stride_d + 1;
    int pdend = min((d + pad_d) / stride_d + 1, pooled_depth);
	
    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width * pooled_depth;
    top_diff += offset;
    if (mask) {
      mask += offset;
      // for (int ph = phstart; ph < phend; ++ph) {
        // for (int pw = pwstart; pw < pwend; ++pw) {
		  // for (int pd = pdstart; pd < pdend; ++pd) {
	   for (int ph = phstart; ph < phend; ph+=k_stride_h) {
        for (int pw = pwstart; pw < pwend; pw+=k_stride_w) {
		  for (int pd = pdstart; pd < pdend; pd+=k_stride_d) {
          if (mask[(ph * pooled_width + pw) * pooled_depth + pd] == (h * width + w) * depth +d) {
            gradient += top_diff[(ph * pooled_width + pw) * pooled_depth + pd];
			}
		  }
        }
      }
    } else {
      top_mask += offset;
      // for (int ph = phstart; ph < phend; ++ph) {
        // for (int pw = pwstart; pw < pwend; ++pw) {
			// for (int pd = pdstart; pd < pdend; ++pd){
	  for (int ph = phstart; ph < phend; ph+=k_stride_h) {
        for (int pw = pwstart; pw < pwend; pw+=k_stride_w) {
		  for (int pd = pdstart; pd < pdend; pd+=k_stride_d) {
			  if (top_mask[(ph * pooled_width + pw) * pooled_depth + pd] == (h * width + w) * depth +d) {
				gradient += top_diff[(ph * pooled_width + pw) * pooled_depth + pd];
			  }
		  }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}
  
  
  

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height, const int width, const int depth, 
	const int pooled_height, const int pooled_width, const int pooled_depth, 
	const int kernel_h, const int kernel_w, const int kernel_d,
    const int ext_kernel_h, const int ext_kernel_w, const int ext_kernel_d,	
	const int stride_h, const int stride_w, const int stride_d, 
	const int kstride_h, const int kstride_w, const int kstride_d,
	const int pad_h, const int pad_w, const int pad_d, Dtype* top_data)
	{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pd = index % pooled_depth;
	int pw = (index / pooled_depth) % pooled_width;
    int ph = (index / pooled_depth / pooled_width ) % pooled_height;
    int c = (index / pooled_depth / pooled_width / pooled_height) % channels;
    int n = index / pooled_depth / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
	int dstart = pd * stride_d - pad_d;
    int hend = min(hstart + ext_kernel_h, height + pad_h);
    int wend = min(wstart + ext_kernel_w, width + pad_w);
	int dend = min(dstart + ext_kernel_d, depth + pad_d);
	int pool_size = (hend - hstart) * (wend - wstart) * (dend - dstart);
	hstart = max(hstart, 0);
    wstart = max(wstart, 0);
	dstart = max(dstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
	dend = min(dend, depth);
    Dtype aveval = 0;
    //bottom_data += (n * channels + c) * height * width;
	bottom_data += (n * channels + c) * height * width * depth;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
		for (int d = dstart; d < dend; ++d){
			aveval += bottom_data[(h * width + w) * depth + d];
		}
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int ext_kernel_h, const int ext_kernel_w,
    const int stride_h, const int stride_w, const int kstride_h, const int kstride_w,
    Dtype* rand_idx, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + ext_kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + ext_kernel_w, width);
    Dtype cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
	
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w,  const int ext_kernel_h, const int ext_kernel_w,
    const int stride_h, const int stride_w, const int kstride_h, const int kstride_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + ext_kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + ext_kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}


template <typename Dtype>
Dtype PoolingSKLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  size_t count = (*top)[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top->size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;

  size_t ext_kernel_h = (kernel_h_ - 1) * kstride_h_ + 1;
  size_t ext_kernel_w = (kernel_w_ - 1) * kstride_w_ + 1;
  size_t ext_kernel_d = (kernel_d_ - 1) * kstride_d_ + 1;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = (*top)[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, depth_, pooled_height_, pooled_width_, pooled_depth_, kernel_h_,
        kernel_w_, kernel_d_, ext_kernel_h, ext_kernel_w, ext_kernel_d,
        stride_h_, stride_w_, stride_d_, kstride_h_, kstride_w_, kstride_d_,
        pad_h_, pad_w_, pad_d_, top_data,
        mask, top_mask);
	//bottom[0]->release_data();
	//bottom[0]->release_cpu_data();
	
	
	if (Caffe::phase() == Caffe::TEST){
		bottom[0]->release_all_data();
		 if (use_top_mask) {
		  (*top)[1]->release_all_data();
		} else {
		  max_idx_.release_all_data();
		}
	  }
	 
   // Dtype* top_data = (*top)[i]->mutable_gpu_data();
    //col_buffer_.release_data();
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, depth_, pooled_height_, pooled_width_, pooled_depth_, kernel_h_,
        kernel_w_, kernel_d_, ext_kernel_h, ext_kernel_w, ext_kernel_d,
        stride_h_, stride_w_, stride_d_, kstride_h_, kstride_w_, kstride_d_,
        pad_h_, pad_w_, pad_d_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    if (Caffe::phase() == Caffe::TRAIN) {
      // We need to create the random index as well.
      caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                            rand_idx_.mutable_gpu_data());
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count),
                                   CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, ext_kernel_h, ext_kernel_w,
          stride_h_, stride_w_, kstride_h_, kstride_w_,
          rand_idx_.mutable_gpu_data(), top_data);
    } else {
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, ext_kernel_h, ext_kernel_w,
          stride_h_, stride_w_, kstride_h_, kstride_w_, top_data);
    }
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
  //return;
  return Dtype(0.);
}

template <typename Dtype>
void PoolingSKLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
	  
	   if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  const int count = (*bottom)[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;	
  int ext_kernel_h = (kernel_h_ - 1) * kstride_h_ + 1;
  int ext_kernel_w = (kernel_w_ - 1) * kstride_w_ + 1;
  int ext_kernel_d = (kernel_d_ - 1) * kstride_d_ + 1;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_idx_.gpu_data();
    }
	

	
	
	
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolBackward_SK<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, top_mask, top[0]->num(), channels_,
        height_, width_, depth_, pooled_height_, pooled_width_, pooled_depth_,
        kernel_h_,kernel_w_,kernel_d_, 
		ext_kernel_h, ext_kernel_w, ext_kernel_d,
		stride_h_,stride_w_,stride_d_, pad_h_, pad_w_, pad_d_,
        kstride_h_, kstride_w_, kstride_d_,
		bottom_diff);
    break;
	 default:
    LOG(FATAL) << "Only support max pooling method for backward propagation...";
  }
  CUDA_POST_KERNEL_CHECK;
  return;
}


INSTANTIATE_CLASS(PoolingSKLayer);


}  // namespace caffe
