#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_max(const size_t num, const size_t channels,
    const size_t spatial_dim, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    size_t n = index / spatial_dim;
    size_t s = index % spatial_dim;
    Dtype maxval = -FLT_MAX;
    for (size_t c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_channel_subtract(const size_t count,
    const size_t num, const size_t channels,
    const size_t spatial_dim, const Dtype* channel_max, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    size_t n = index / channels / spatial_dim;
    size_t s = index % spatial_dim;
    data[index] -= channel_max[n * spatial_dim + s];
  }
}


template <typename Dtype>
__global__ void kernel_channel_sum(const size_t num, const size_t channels,
    const size_t spatial_dim, const Dtype* data, Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    size_t n = index / spatial_dim;
    size_t s = index % spatial_dim;
    Dtype sum = 0;
    for (size_t c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void kernel_channel_div(const size_t count,
    const size_t num, const size_t channels,
    const size_t spatial_dim, const Dtype* channel_sum, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    size_t n = index / channels / spatial_dim;
    size_t s = index % spatial_dim;
    data[index] /= channel_sum[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_channel_dot(const size_t num, const size_t channels,
    const size_t spatial_dim, const Dtype* data_1, const Dtype* data_2,
    Dtype* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype dot = 0;
    for (size_t c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}




template <typename Dtype>
__global__ void kernel_get_max(const int num, const int dim,
    const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num) {
    Dtype maxval = -FLT_MAX;
    for (int i = 0; i < dim; ++i) {
      maxval = max(data[index * dim + i], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_softmax_div(const int num, const int dim,
    const Dtype* scale, Dtype* data) {
  CUDA_KERNEL_LOOP(index, num * dim) {
    int n = index / dim;
    data[index] /= scale[n];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int num, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
Dtype SoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
	//LOG(INFO)<<"SoftmaxLayer forward_pgu ";
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  
  size_t count = bottom[0]->count();
  size_t channels = (*top)[0]->channels();
  size_t num = bottom[0]->num();
  size_t dim = bottom[0]->count() / bottom[0]->num();
  caffe_copy(bottom[0]->count(), bottom_data, top_data);
  
  kernel_channel_max<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
	  
  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
  // exponentiate
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_data, top_data);
  // sum after exp
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // divide
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
	  
  
  
  // // we need to subtract the max to avoid numerical issues, compute the exp,
  // // and then normalize.
  // // Compute max
  // // NOLINT_NEXT_LINE(whitespace/operators)
  // kernel_get_max<Dtype><<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS>>>(
      // num, dim, bottom_data, scale_data);
  // // subtraction
  // caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, -1.,
      // scale_data, sum_multiplier_.gpu_data(), 1., top_data);
  // // Perform exponentiation
  // // NOLINT_NEXT_LINE(whitespace/operators)
  // kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(num * dim), CAFFE_CUDA_NUM_THREADS>>>(
      // num * dim, top_data, top_data);
  // // sum after exp
  // caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, 1., top_data,
      // sum_multiplier_.gpu_data(), 0., scale_data);
  // // Do division
  // // NOLINT_NEXT_LINE(whitespace/operators)
  // kernel_softmax_div<Dtype><<<CAFFE_GET_BLOCKS(num * dim),
                              // CAFFE_CUDA_NUM_THREADS>>>(
      // num, dim, scale_data, top_data);
  //LOG(INFO)<<"Done Softmax GPU...";
  return Dtype(0);
}

// TODO(Yangqing): implement the GPU version of softmax.
template <typename Dtype>
void SoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
	LOG(INFO)<<"SoftmaxLayer backward_pgu ";
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  Dtype* scale_data = scale_.mutable_gpu_data();
  size_t num = top[0]->num();
  size_t count = top[0]->count();
  size_t channels = top[0]->channels();
  //int dim = top[0]->count() / top[0]->num();
  size_t dim = top[0]->count() / top[0]->num();
  caffe_copy(top[0]->count(), top_diff, bottom_diff);
  
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_dot<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_,
      top_diff, top_data, scale_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, bottom_diff);
  // elementwise multiplication
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
  
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff
  // cuda dot returns the result to cpu, so we temporarily change the pointer
  // mode
  // CUBLAS_CHECK(hipblasSetPointerMode(Caffe::cublas_handle(),
      // HIPBLAS_POINTER_MODE_DEVICE));
  // Dtype* scale_data = scale_.mutable_gpu_data();
  // for (int i = 0; i < num; ++i) {
    // caffe_gpu_dot<Dtype>(dim, top_diff + i * dim,
        // top_data + i * dim, scale_data + i);
  // }
  // CUBLAS_CHECK(hipblasSetPointerMode(Caffe::cublas_handle(),
      // HIPBLAS_POINTER_MODE_HOST));
  // // subtraction
  // caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, -1.,
      // scale_.gpu_data(), sum_multiplier_.gpu_data(), 1., bottom_diff);
  // // elementwise multiplication
  // caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
  LOG(INFO)<<"done SoftmaxLayer backward_gpu ";
}

INSTANTIATE_CLASS(SoftmaxLayer);


}  // namespace caffe
