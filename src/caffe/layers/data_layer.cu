// Copyright 2014 BVLC and contributors.

#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;

namespace caffe {

template <typename Dtype>
Dtype DataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  caffe_copy(prefetch_data_->count(), prefetch_data_->cpu_data(),
      (*top)[0]->mutable_gpu_data());
  //LOG(INFO)<<"data layer data copyed";
  if (output_labels_) {
    caffe_copy(prefetch_label_->count(), prefetch_label_->cpu_data(),
        (*top)[1]->mutable_gpu_data());
		//const Dtype* top_label = (*top)[1]->mutable_gpu_data();
		//LOG(INFO)<<top_label[0];
	//LOG(INFO)<<"label layer  copied";
  }
  // Start a new prefetch thread
  
  CreatePrefetchThread();
  //LOG(INFO)<<"start prefettch";
  return Dtype(0.);
}

INSTANTIATE_CLASS(DataLayer);

}  // namespace caffe
